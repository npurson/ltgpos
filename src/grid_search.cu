#include "hip/hip_runtime.h"
#include "grid_search.h"


__global__ void calGirdGoodness2d_G(ssrinfo_t sinfo, grdinfo_t ginfo)
{
    int num_ssrs = sinfo.num_ssrs;
    long involved = sinfo.involved;
    double* ssr_locs = sinfo.ssr_locs;
    double* ssr_times = sinfo.ssr_times;

    double x = ginfo.sch_dom[0] + ginfo.grd_inv[0] * threadIdx.x;
    double y = ginfo.sch_dom[2] + ginfo.grd_inv[1] * blockIdx.x;
    double dt[kMaxNumSsrs];
    double t0 = 0, err = 0;

    int num_involved = 0;
    for (int i = 0; i < num_ssrs; i++) {
        if (!(involved & mask << i)) continue;
        ++num_involved;
        dt[i] = getGeoDistance2d_D(ssr_locs[i * 2], ssr_locs[i * 2 + 1], x, y) / C;
        // Is referrence sensor
        // if (involved & -involved & mask << i) { t0 = dt[i]; continue; }
        t0 += dt[i] - ssr_times[i];
    }
    t0 /= num_involved;

    for (int i = 0; i < num_ssrs; i++) {
        if (!(involved & mask << i)) continue;
        dt[i] -= t0 + ssr_times[i];
        err += dt[i] * dt[i] * 1e6;
    }
    err /= num_involved - 1;
    ginfo.douts[blockIdx.x * blockDim.x + threadIdx.x] = err;
}


void grid_search(ssrinfo_t* ssrinfo, grdinfo_t* grdinfo, schdata_t* schdata)
{
    if (get_num_involved(schdata->involved) < 3) {
        // fprintf(stderr, "%s(%d): grid search expects number of involved sensors >= 3, but got %d.\n",
        //         __FILE__, __LINE__, get_num_involved(schdata->involved));
        schdata->out_ans[4] = INFINITY;
        return;
    }
    ssrinfo->num_ssrs = schdata->num_ssrs;
    ssrinfo->involved = schdata->involved;
    double* ssr_locs = schdata->ssr_locs;
    double* ssr_times = schdata->ssr_times;
    int num_ssrs = ssrinfo->num_ssrs;
    hipMemcpy(ssrinfo->ssr_locs, ssr_locs, num_ssrs * 2 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(ssrinfo->ssr_times, ssr_times, num_ssrs * sizeof(double), hipMemcpyHostToDevice);

    double* out_ans = schdata->out_ans;
    double* sch_dom = grdinfo->sch_dom;
    double* grd_inv = grdinfo->grd_inv;
    double* houts   = grdinfo->houts;
    double min_err;
    int min_idx = 0;

    for (int i = 0; i < kNumSchs; i++) {
        if (!i) {
            // memcpy(sch_dom, schdata->sch_dom, 4 * sizeof(double));
            gen_sch_dom(ssr_locs, num_ssrs, schdata->involved, sch_dom);
        } else {
            // Generate search domain based on result of previous search.
            for (int j = 0; j < 4; j++) {
                sch_dom[j] = out_ans[j / 2 + 1] + grd_inv[j / 2] * kNumNxtSchInvs * ((j % 2) ? 1 : -1);
            }
        }
        for (int j = 0; j < 2; j++) {
            grd_inv[j] = (sch_dom[j * 2 + 1] - sch_dom[j * 2]) / (kMaxGrdSize - 1);
        }

        dim3 grid(kMaxGrdSize), block(kMaxGrdSize);
        calGirdGoodness2d_G <<<grid, block>>> (*ssrinfo, *grdinfo);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "%s(%d): %s.\n", __FILE__, __LINE__, hipGetErrorString(err));
            out_ans[4] = INFINITY;
            return;
        }
        hipMemcpy(houts, grdinfo->douts, kMaxGrdNum * sizeof(double), hipMemcpyDeviceToHost);

        min_err = houts[0];
        min_idx = 0;
        for (int j = 1; j < kMaxGrdNum; j++) {
            if (houts[j] >= min_err) continue;
            min_err = houts[j];
            min_idx = j;
        }
        out_ans[1] = sch_dom[0] + min_idx % kMaxGrdSize * grd_inv[0];
        out_ans[2] = sch_dom[2] + min_idx / kMaxGrdSize % kMaxGrdSize * grd_inv[1];

        #ifdef PLT
        FILE* fp = fopen("figures/grdres.txt", "a");
        for (int j = 0; j < kMaxGrdNum; j++) {
            fprintf(fp, "%f ", houts[j]);
        }
        fprintf(fp, "%f %f %f %f %f %f\n", sch_dom[0], sch_dom[1], sch_dom[2], sch_dom[3], out_ans[1], out_ans[2]);
        fclose(fp);
        #endif
    }
    int ref_idx = get_first_involved(schdata->involved);
    out_ans[0] = ssr_times[ref_idx] - getGeoDistance2d_H(ssr_locs[ref_idx * 2], ssr_locs[ref_idx * 2 + 1], out_ans[1], out_ans[2]) / C;
    out_ans[4] = min_err;
    return;
}
