#include "hip/hip_runtime.h"
#include "grid_search.h"


__constant__ __device__ long dmask = 0x1;


__global__ void calGirdGoodness2d_G(ssrinfo_t sinfo, grdinfo_t ginfo)
{
    int num_dims = sinfo.is3d ? 3 : 2;
    int num_ssrs = sinfo.num_ssrs;
    long involved = sinfo.involved;
    F* ssr_locs = sinfo.ssr_locs;
    F* ssr_times = sinfo.ssr_times;

    F x = ginfo.sch_dom[0] + ginfo.grd_inv[0] * threadIdx.x;
    F y = ginfo.sch_dom[2] + ginfo.grd_inv[1] * blockIdx.x;
    F t0, dt, err = 0;

    for (int i = 0; i < num_ssrs; i++) {
        if (!(involved & dmask << i)) continue;
        dt = getGeoDistance2d_D(ssr_locs[i * num_dims], ssr_locs[i * num_dims + 1], x, y) / C;

        if (involved & -involved & dmask << i) { t0 = dt; continue; }  // Is referrence sensor
        dt -= t0 + ssr_times[i];
        err += dt * dt * 1e6;
    }
    err /= num_ssrs - 2;
    ginfo.douts[blockIdx.x * blockDim.x + threadIdx.x] = err;
}


__global__ void calGirdGoodness3d_G(ssrinfo_t sinfo, grdinfo_t ginfo)
{
    int num_ssrs = sinfo.num_ssrs;
    long involved = sinfo.involved;
    F* ssr_locs = sinfo.ssr_locs;
    F* ssr_times = sinfo.ssr_times;

    F x = ginfo.sch_dom[0] + ginfo.grd_inv[0] * threadIdx.x;
    F y = ginfo.sch_dom[2] + ginfo.grd_inv[1] * blockIdx.x;
    F z = ginfo.sch_dom[3] + ginfo.grd_inv[2] * blockIdx.y;
    F t0, dt, err = 0;

    for (int i = 0; i < num_ssrs; i++) {
        if (!(involved & dmask << i)) continue;
        dt = getGeoDistance3d_D(ssr_locs[i*3], ssr_locs[i*3+1], ssr_locs[i*3+2], x, y, z) / C;

        if (involved & -involved & dmask << i) { t0 = dt; continue; }  // Is referrence sensor
        dt -= t0 + ssr_times[i];
        err += dt * dt * 1e6;
    }
    err /= num_ssrs - 2;
    ginfo.douts[blockIdx.y * blockDim.x * gridDim.x +
                blockIdx.x * blockDim.x + threadIdx.x] = err;
}


// int dump_to_file(F* outs, int grid_sizes[3], const char* filename)
// {
//     FILE* fp = NULL;
//     fp = fopen(filename, "w");
//     if (!fp) {
//         fprintf(stderr, "%s(%d): failed to open file %s.\n", __FILE__, __LINE__, filename);
//         return 1;
//     }

//     for (int i = 0; i < grid_sizes[2]; i++) {
//         for (int j = 0; j < grid_sizes[1]; j++) {
//             for (int k = 0; k < grid_sizes[0]; k++) {
//                 fprintf(fp, "%8.2f ", outs[i * grid_sizes[1] * grid_sizes[0] + j * grid_sizes[1] + k]);
//             }
//             fprintf(fp, "\n");
//         }
//         fprintf(fp, "\n");
//     }
//     fclose(fp);
//     printf("[Dump] outputs dumped to %s\n", filename);
//     return 0;
// }


void grid_search(ssrinfo_t* sinfos, grdinfo_t* ginfos, schdata_t* schdata)
{
    ssrinfo_t* ssrinfo;
    grdinfo_t* grdinfo;

    int num_ssrs = schdata->num_ssrs;
    int num_dims = schdata->is3d ? 3 : 2;
    F* ssr_locs  = schdata->ssr_locs;
    F* ssr_times = schdata->ssr_times;
    F* out_ans   = schdata->out_ans;

    int grd_size, grd_sizes[3];
    F* sch_dom, * grd_inv, * houts, min_err;
    bool is3d;

    for (int i = 0; i < kNumSchs; i++) {
        ssrinfo = &sinfos[i];
        grdinfo = &ginfos[i];

        sch_dom = grdinfo->sch_dom;
        grd_inv = grdinfo->grd_inv;
        houts = grdinfo->houts;
        is3d = schdata->is3d && i;

        // Initialize search domain.
        if (!i) memcpy(sch_dom, schdata->sch_dom, 6 * sizeof(F));
        // Generate search domain based on result of previous search.
        else {
            for (int j = 0; j < 4; j++) {
                sch_dom[j] = out_ans[j/2+1] + grd_inv[j/2] * kNxtSchDomInvs * ((j % 2) ? 1 : -1);
            }
        }
        // Do 3D search in height of 0 ~ 20 km.
        sch_dom[4] = 0;
        sch_dom[5] = is3d ? 20 : 0;

        for (int j = 0; j < 3; j++) {
            grd_inv[j] = (j == 2) ? 1 : max((sch_dom[j*2+1] - sch_dom[j*2]) / (gMaxGridNum - 1) * (is3d ? 5 : 1.02), 2e-5);
            grd_sizes[j] = (sch_dom[j*2+1] - sch_dom[j*2]) / grd_inv[j] + 1;
        }
        grd_size = grd_sizes[0] * grd_sizes[1] * grd_sizes[2];

        ssrinfo->num_ssrs = schdata->num_ssrs;
        ssrinfo->involved = schdata->involved;
        ssrinfo->is3d = schdata->is3d;
        hipMemcpy(ssrinfo->ssr_locs, ssr_locs, num_ssrs * num_dims * sizeof(F), hipMemcpyHostToDevice);
        hipMemcpy(ssrinfo->ssr_times, ssr_times, num_ssrs * sizeof(F), hipMemcpyHostToDevice);

        if (is3d) {
            dim3 grid(grd_sizes[1], grd_sizes[2]), block(grd_sizes[0]);
            calGirdGoodness3d_G <<<grid, block>>> (*ssrinfo, *grdinfo);
        } else {
            dim3 grid(grd_sizes[1]), block(grd_sizes[0]);
            calGirdGoodness2d_G <<<grid, block>>> (*ssrinfo, *grdinfo);
        }
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "%s(%d): %s.\n", __FILE__, __LINE__, hipGetErrorString(err));
            out_ans[4] = -1;
            return;
        }
        hipMemcpy(houts, grdinfo->douts, grd_size * sizeof(F), hipMemcpyDeviceToHost);

        min_err = houts[0];
        int min_idx = 0;
        for (int j = 1; j < grd_size; j++) {
            if (houts[j] >= min_err) continue;
            min_err = houts[j];
            min_idx = j;
        }
        // dump_to_file(houts, grd_sizes, "test/gridres.csv");

        out_ans[1] = sch_dom[0] + min_idx % grd_sizes[0] * grd_inv[0];
        out_ans[2] = sch_dom[2] + min_idx / grd_sizes[0] % grd_sizes[1] * grd_inv[1];
        out_ans[3] = sch_dom[4] + min_idx / grd_sizes[0] / grd_sizes[1] * grd_inv[2];
    }

    int ref_idx = log2(schdata->involved);
    out_ans[0] = ssr_times[ref_idx] - schdata->is3d ?
                 getGeoDistance3d_H(ssr_locs[ref_idx * 3], ssr_locs[ref_idx * 3 + 1], ssr_locs[ref_idx * 3 + 2], out_ans[1], out_ans[2], out_ans[3]) / C :
                 getGeoDistance2d_H(ssr_locs[ref_idx * 2], ssr_locs[ref_idx * 2 + 1],out_ans[1], out_ans[2]) / C;
    out_ans[4] = min_err;
    return;
}
