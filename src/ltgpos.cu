#include "ltgpos.h"


ssrinfo_t gSsrInfo;
grdinfo_t gGrdInfo;
bool isInit = false;


int initSysInfo()
{
    if (malloc_s(&gGrdInfo.houts, kMaxGrdNum)) return 1;
    if (cudaMalloc_s(&gGrdInfo.douts, kMaxGrdNum)) return 1;
    if (cudaMalloc_s(&gSsrInfo.ssr_locs, kMaxNumSsrs * 2)) return 1;
    if (cudaMalloc_s(&gSsrInfo.ssr_times, kMaxNumSsrs)) return 1;
    isInit = true;
    return 0;
}


void freeSysInfo()
{
    free(gGrdInfo.houts);
    hipFree(gGrdInfo.douts);
    hipFree(gSsrInfo.ssr_locs);
    hipFree(gSsrInfo.ssr_times);
    isInit = false;
}


// Divide & conquer.
long dac_search(schdata_t& d)
{
    long involved = 0, prev_involved;
    // printf("%lx\n", d.involved);
    std::vector<long> combs = comb_mapper(d.involved);
    for (int i = 0; i != combs.size(); i++) {
        // printf("sub: %lx\n", combs[i]);
        d.involved = combs[i];
        grid_search(&gSsrInfo, &gGrdInfo, &d);
        // double* out_ans = d.out_ans;
        // printf("%7.4lf  %8.4lf  %.4lf\n", out_ans[1], out_ans[2], out_ans[4]);
        prev_involved = involved;
        involved |= d.out_ans[4] > kGoodThres ? dac_search(d) : combs[i];
        d.involved = involved;
        grid_search(&gSsrInfo, &gGrdInfo, &d);
        if (d.out_ans[4] > kGoodThres) involved = prev_involved;
    }
    // printf("fin: %lx\n", involved);
    return involved;
}


char* ltgpos(char* str)
{
    if (!isInit && initSysInfo()) {
        fprintf(stderr, "%s(%d): failed to initialize sysinfo.\n", __FILE__, __LINE__);
        return NULL;
    }

    schdata_t schdata;
    // Ensure jarr is deleted before return.
    cJSON* jarr = parseJsonStr(str, &schdata);
    if (!jarr) return NULL;

    grid_search(&gSsrInfo, &gGrdInfo, &schdata);
    if (schdata.out_ans[4] >= kGoodThres) {
        long prev_involved = schdata.involved;
        long involved = dac_search(schdata);
        schdata.involved = involved ? involved : prev_involved;
        grid_search(&gSsrInfo, &gGrdInfo, &schdata);
    }

    #ifdef TEST
    double* out_ans = schdata.out_ans;
    // double* sch_dom = schdata.sch_dom;
    // double* ssr_locs = schdata.ssr_locs;
    printf("%7.4lf  %8.4lf  %.4lf\n", out_ans[1], out_ans[2], out_ans[4]);
    // printf("%7.4lf  %7.4lf  %8.4lf %8.4lf\n", sch_dom[0], sch_dom[1], sch_dom[2], sch_dom[3]);
    // printf("%.2lf\n", (sch_dom[1] - sch_dom[0]) * (sch_dom[3] - sch_dom[2]));
    // printf("%d\n", schdata.num_ssrs);
    // for (int i = 0; i < schdata.num_ssrs; i++) printf("%.4lf, %.4lf\n", ssr_locs[i * 2], ssr_locs[i * 2 + 1]);
    #endif

    // Ensure the string returned is deallocated after use.
    return formatRetJsonStr(&schdata, jarr);
}
