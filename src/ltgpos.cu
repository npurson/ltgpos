#include "ltgpos.h"


ssrinfo_t gSsrInfos[kNumSchs];
grdinfo_t gGrdInfos[kNumSchs];
bool isInit = false;


int initSysInfo()
{
    if (malloc_s(&gGrdInfos[0].houts, gMaxGridSize)) return 1;
    gGrdInfos[1].houts = gGrdInfos[0].houts;
    if (cudaMalloc_s(&gGrdInfos[0].douts, gMaxGridSize)) return 1;
    gGrdInfos[1].douts = gGrdInfos[0].douts;

    if (cudaMalloc_s(&gSsrInfos[0].ssr_locs, kMaxNumSsrs * 2)) return 1;
    gSsrInfos[1].ssr_locs = gSsrInfos[0].ssr_locs;
    if (cudaMalloc_s(&gSsrInfos[0].ssr_times, kMaxNumSsrs)) return 1;
    gSsrInfos[1].ssr_times = gSsrInfos[0].ssr_times;

    isInit = true;
    return 0;
}


void freeSysInfo()
{
    free(gGrdInfos[0].houts);
    hipFree(gGrdInfos[0].douts);
    hipFree(gSsrInfos[0].ssr_locs);
    hipFree(gSsrInfos[0].ssr_times);
    isInit = false;
}


// Divide & conquer.
long dac_search(schdata_t& d)
{
    long involved = 0, prev_involved;
    // printf("%lx\n", d.involved);
    std::vector<long> combs = comb_mapper(d.involved);
    for (int i = 0; i != combs.size(); i++) {
        // printf("sub: %lx\n", combs[i]);
        d.involved = combs[i];
        grid_search(gSsrInfos, gGrdInfos, &d);
        // F* out_ans = d.out_ans;
        // printf("%7.4lf  %8.4lf  %.4lf\n", out_ans[1], out_ans[2], out_ans[4]);
        prev_involved = involved;
        involved |= d.out_ans[4] > gGoodThres ? dac_search(d) : combs[i];
        d.involved = involved;
        grid_search(gSsrInfos, gGrdInfos, &d);
        if (d.out_ans[4] > gGoodThres) involved = prev_involved;
    }
    // printf("fin: %lx\n", involved);
    return involved;
}


char* ltgpos(char* str)
{
    if (!isInit && initSysInfo()) {
        fprintf(stderr, "%s(%d): failed to initialize sysinfo.\n", __FILE__, __LINE__);
        return NULL;
    }

    schdata_t schdata;
    F* out_ans = schdata.out_ans;

    // Ensure jarr is deleted before return.
    cJSON* jarr = parseJsonStr(str, &schdata);
    if (!jarr) return NULL;

    grid_search(gSsrInfos, gGrdInfos, &schdata);
    if (out_ans[4] >= gGoodThres) {
        long prev_involved = schdata.involved;
        long involved = dac_search(schdata);
        schdata.involved = involved ? involved : prev_involved;
        grid_search(gSsrInfos, gGrdInfos, &schdata);
    }

    #ifdef TEST
    // F* sch_dom = schdata.sch_dom;
    // F* ssr_locs = schdata.ssr_locs;
    printf("%7.4lf  %8.4lf  %.4lf\n", out_ans[1], out_ans[2], out_ans[4]);
    // printf("%7.4lf  %7.4lf  %8.4lf %8.4lf\n", sch_dom[0], sch_dom[1], sch_dom[2], sch_dom[3]);
    // printf("%d\n", schdata.num_ssrs);
    // for (int i = 0; i < schdata.num_ssrs; i++) printf("%.4lf, %.4lf\n", ssr_locs[i * 2], ssr_locs[i * 2 + 1]);
    #endif

    // Ensure the string returned is deallocated after use.
    return formatRetJsonStr(&schdata, jarr);
}
